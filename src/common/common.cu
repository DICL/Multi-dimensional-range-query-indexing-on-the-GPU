#include "hip/hip_runtime.h"
#include <common.h>


void profileCopies(float        *h_a,
		float        *h_b,
		float        *d,
		unsigned int  n,
		char         *desc)
{
	printf("\n%s transfers\n", desc);

	unsigned int bytes = n * sizeof(float);

	// events for timing
	hipEvent_t startEvent, stopEvent;

	checkCuda( hipEventCreate(&startEvent) );
	checkCuda( hipEventCreate(&stopEvent) );

	checkCuda( hipEventRecord(startEvent, 0) );
	checkCuda( hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice) );
	checkCuda( hipEventRecord(stopEvent, 0) );
	checkCuda( hipEventSynchronize(stopEvent) );

	float time;
	checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
	printf("  Host to Device bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

	checkCuda( hipEventRecord(startEvent, 0) );
	checkCuda( hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost) );
	checkCuda( hipEventRecord(stopEvent, 0) );
	checkCuda( hipEventSynchronize(stopEvent) );

	checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
	printf("  Device to Host bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

	for (int i = 0; i < n; ++i) {
		if (h_a[i] != h_b[i]) {
			printf("*** %s transfers failed ***", desc);
			break;
		}
	}
	checkCuda( hipEventDestroy(startEvent) );
	checkCuda( hipEventDestroy(stopEvent) );
}



void printHelp(char **argv)
{
	std::cerr << "Usage:\n" << *argv << endl << 
			" -d number of data\n" 
			" [ -q number of queries, default : 0 (insertion mode) ]\n" 
			" [ -i index type, default : R-trees]\n"
			" [ -m search algorithm type, 1: MPES, 2: MPTS, 3: MPHR, 4: MPHR2\n \
			                            5: Short-Stack, 6: Parent-Link, 7: Skip-Pointer ]\n"
			" [ -o distribution policy, default : braided version]\n"
			" [ -b braided version, number of block, default : 128 ]\n"
			" [ -p partitioned version, number of block ]\n" 
			" [ -s selection ratio(%), default : 1 (%) ]\n"
			" [ -c number of cpu cores, default : 1 ]\n" 
			" [ -w workload offset, default : 0 ]\n" 
			" [ -v Specified device(GPU) id, default : 0 ]\n" 
			"\n e.g:   ./cuda -d 1000000 -q 1000 -s 0.5 -c 4 -w 3\n" 
			<< std::endl;
}

bool ParseArgs(int argc, char **argv) 
{

	//!!! add the policy (eg., round robin, dema, etc..)

	//assign default values
	NUMDATA = 0;
	NUMSEARCH = 0;
	PARTITIONED = 1; // 1 : braided version, over than 1 : partitioned version
	NUMBLOCK = 128; 
	strcpy(SELECTIVITY , "0.01"); // 0.05 %
	NCPUCORES =  0;
	WORKLOAD = 0;
	POLICY = 0;
	BUILD_TYPE = 0;
	DEVICE_ID = 0;
	keepDoing = 0;

	for( int i=0; i<7; i++)
	{
		t_time[i] = 0.0f;
		t_visit[i] = .0f;
		t_rootVisit[i] = .0f;
		t_pop[i] = .0f;
		t_push[i] = .0f;
		t_parent[i] = .0f;
		t_skipPointer[i] = .0f;
		METHOD[i] = false;
	}
	METHOD[7] = false;

	static const char *options="d:D:q:Q:p:P:b:B:s:S:c:C:w:W:o:O:i:I:m:M:k:K:v:V:";
	extern char *optarg;
	int c;
	char char_NUMDATA[20];

	while ((c = getopt(argc, argv, options)) != -1) {
		switch (c) {
			case 'v':
			case 'V': DEVICE_ID = atoi(optarg); break;
			case 'k':
			case 'K': keepDoing = atoi(optarg); break;
			case 'd':
			case 'D': strcpy(char_NUMDATA,optarg); break;
			case 'q':
			case 'Q': NUMSEARCH = atoi(optarg); break;
			case 'p':
			case 'P': NUMBLOCK = PARTITIONED = atoi(optarg); break;
			case 'b':
			case 'B': NUMBLOCK = atoi(optarg); PARTITIONED=1; break;
			case 's':
			case 'S': strcpy(SELECTIVITY , optarg); break;
			case 'c':
			case 'C': NCPUCORES = atoi(optarg); break;
			case 'w':
			case 'W': WORKLOAD = atoi(optarg); break;
			case 'o':
			case 'O': POLICY = atoi(optarg); break;
			case 'i':
			case 'I': BUILD_TYPE = atoi(optarg); break;
			case 'm':
			case 'M': METHOD[atoi(optarg)-1] = true;
								optind--;
								for( ;optind < argc && *argv[optind] != '-'; optind++)
								{
									METHOD[atoi( argv[optind] )-1] = true;
								}
								break;
			default: break;
		} // end of switch
	} // end of while

	if( NCPUCORES > 0 )
		NCPUCORES = ( PARTITIONED > 1 ) ? PARTITIONED : 1;  

	ch_root = (char**)malloc(PARTITIONED*sizeof(char*));
	cd_root = (char**)malloc(PARTITIONED*sizeof(char*));


	if( char_NUMDATA[strlen(char_NUMDATA)-1] == 'm')
	{
    char_NUMDATA[strlen(char_NUMDATA)-1] =  '\n';
		NUMDATA = atoi(char_NUMDATA)*1000000;
	}
	else 
		NUMDATA = atoi(char_NUMDATA);

	if( NUMDATA <= 1000000 )
		strcpy( querySize,"1m");
	else if( NUMDATA == 2000000)
		strcpy( querySize,"2m");
	else 	if( NUMDATA == 4000000)
		strcpy( querySize,"4m");
	else 	if( NUMDATA == 8000000)
		strcpy( querySize,"8m");
	else
		strcpy( querySize,"16m");

	/*	else 	if( NUMDATA == 16000000 )
		strcpy( querySize,"16m");
	else 	if( NUMDATA == 32000000)
		strcpy( querySize,"32m");
	else 	if( NUMDATA == 40000000)
		strcpy( querySize,"40m");
	else
		strcpy( querySize,"1m");
		*/





	if( METHOD[7] == true)
		METHOD[0] = METHOD[1] = METHOD[2] =  METHOD[3] = METHOD[4] = METHOD[5] = METHOD[6] = true;


	if( (METHOD[0] || METHOD[1] || METHOD[2] ||  METHOD[3] || METHOD[4] || METHOD[5] || METHOD[6]) && NUMSEARCH == 0)
	{
		NUMSEARCH = 1000;
	}



	if(BUILD_TYPE == 0)
	{
		printf("DATATYPE : %s, PGSIZE %d, NUMDIMS %d, NUMBLOCK %d, NUMTHREADS %d,  NODECARD %d, NUMDATA %d, NUMSEARCH %d, SELECTION RATIO %s, NCPU %d,   PARTITIONED %d, ",
				    DATATYPE,      PGSIZE,    NUMDIMS,    NUMBLOCK,    NUMTHREADS,     NODECARD,    NUMDATA,    NUMSEARCH,    SELECTIVITY ,       NCPUCORES, PARTITIONED       );
	}
	else
	{
		printf("DATATYPE : %s, PGSIZE %d, NUMDIMS %d, NUMBLOCK %d, NUMTHREADS %d,  NODECARD %d, NUMDATA %d, NUMSEARCH %d, SELECTION RATIO %s, NCPU %d,   PARTITIONED %d, ",
	    		  DATATYPE,      BVH_PGSIZE,    NUMDIMS,    NUMBLOCK,    NUMTHREADS,     NODECARD,    NUMDATA,    NUMSEARCH,    SELECTIVITY ,       NCPUCORES, PARTITIONED       );
	}

	if( BUILD_TYPE == 0 )
		printf("\n\nRTrees will be build up.. \n");
	else if ( BUILD_TYPE == 1 || BUILD_TYPE == 2 )
		printf("\n\nBVH-Trees (TYPE : %d )  will be build up..\n", BUILD_TYPE);
	else
		printf("\n\nHilbertRadix Tree (TYPE : %d )  will be build up..\n", BUILD_TYPE);


	if( POLICY == 0 )
		printf("Original distribution\n");
	else
		printf("Roundrobin distribution\n");


	return true;
}



bool  RectOverlap(struct Rect *r, struct Rect *s)
{
	int i, j;

	for (i=0; i<NUMDIMS; i++)
	{
		j = i + NUMDIMS;  

		if (r->boundary[i] > s->boundary[j] || s->boundary[i] > r->boundary[j])
		{
			return false;
		}
	}
	return true;
}
__device__ bool devRectOverlap(struct Rect *r, struct Rect *s)
{
	int i,j;

	for(i=0; i<NUMDIMS; i++){
		j = i + NUMDIMS;
		if(r->boundary[i] > s->boundary[j] || s->boundary[i] > r->boundary[j]){
			return false;
		}
	}
	return true;
}
__device__ bool dev_Node_SOA_Overlap(struct Rect *r, struct Node_SOA* n, int tid)
{
	int i,j, off1, off2;

	for(i=0; i<NUMDIMS; i++){
		j = i + NUMDIMS;
		off1 = i*NODECARD;
		off2 = j*NODECARD;

		if(r->boundary[i] > n->boundary[off2+tid] || n->boundary[off1+tid] > r->boundary[j]){
			return false;
		}
	}
	return true;
}
__device__ bool dev_BVH_Node_SOA_Overlap(struct Rect *r, BVH_Node_SOA* n, int tid)
{
	int i,j, off1, off2;


	for(i=0; i<NUMDIMS; i++){
		j = i + NUMDIMS;
		off1 = i*NODECARD;
		off2 = j*NODECARD;

		if(r->boundary[i] > n->boundary[off2+tid] || n->boundary[off1+tid] > r->boundary[j]){
			return false;
		}
	}
	return true;
}
bool RadixNode_SOA_Overlap(struct Rect *r, RadixTreeNode_SOA* n, int tid)
{
	int i, j, min, max;

	for(i=0; i<NUMDIMS; i++){
		j = i+NUMDIMS;
		min = NODECARD*(2*i+tid%2);
		max = NODECARD*(2*j+tid%2);

			//printf("tid %d r boundary[%d] %f > n boundary[%d] %f , n boundary[%d] %f > r boundary[%d] %f \n", 
			//		    tid, i, max+tid/2, min+tid/2, j,r->boundary[i], n->boundary[max+tid/2], n->boundary[min+tid/2], r->boundary[j]);
		if(r->boundary[i] > n->boundary[max+tid/2] || n->boundary[min+tid/2] > r->boundary[j]){
			return false;
		}
	}
	return true;
}

__device__ bool dev_RadixNode_SOA_Overlap(struct Rect *r, RadixTreeNode_SOA* n, int tid)
{
	int i,j, min, max;

	/*
	if( tid == 0 )
	{
		printf("r %f r %f r %f\n",r->boundary[0], r->boundary[1],r->boundary[2]);
		printf("r %f r %f r %f\n",r->boundary[3], r->boundary[4],r->boundary[5]);

		printf("n %f n %f n %f\n",n->boundary[0], n->boundary[NODECARD*2],n->boundary[NODECARD*4]);
		printf("n %f n %f n %f\n",n->boundary[NODECARD*6], n->boundary[NODECARD*8],n->boundary[NODECARD*10]);

		printf("n index %llu\n",n->index[0]);
	}
	__syncthreads();
	*/

	for(i=0; i<NUMDIMS; i++){
		j = i+NUMDIMS;
		min = NODECARD*(2*i+tid%2);
		max = NODECARD*(2*j+tid%2);

		if(r->boundary[i] > n->boundary[max+tid/2] || n->boundary[min+tid/2] > r->boundary[j]){
			return false;
		}
	}
	return true;
}

__device__ bool dev_Node_SOA_Overlap2(struct Rect *r, struct Node_SOA* n)
{
	int tid = threadIdx.x;
	int i,j, off1, off2;

	for(i=0; i<NUMDIMS; i++){
		j = i + NUMDIMS;
		off1 = i*NODECARD;
		off2 = j*NODECARD;

		if(r->boundary[i] > n->boundary[off2+tid] || n->boundary[off1+tid] > r->boundary[j]){
			return false;
		}
	}
	return true;
}
__device__ bool dev_BVH_Node_SOA_Overlap2(struct Rect *r, BVH_Node_SOA* n)
{
	int tid = threadIdx.x;
	int i,j, off1, off2;


	for(i=0; i<NUMDIMS; i++){
		j = i + NUMDIMS;
		off1 = i*NODECARD;
		off2 = j*NODECARD;

		if(r->boundary[i] > n->boundary[off2+tid] || n->boundary[off1+tid] > r->boundary[j]){
			return false;
		}
	}
	return true;
}

float IntersectedRectArea(struct Rect *r1, struct Rect *r2)
{
	int i,j;

	float area = 1.0f;

	for( i=0; i<NUMDIMS; i++)
	{
		j=i+NUMDIMS;
		area *= min( r1->boundary[j], r2->boundary[j])-max(r1->boundary[i], r2->boundary[i]);
	}
	return area;
}

void checkNodeOverlap(BVH_Node *n, float* area)
{
	if (n->level > 0) // this is an internal node in the tree //
	{
		for(int i1 = 0; i1<n->count-1; i1++)
			for(int i2 = i1+1; i2<n->count; i2++)
				if( RectOverlap(&n->branch[i1].rect, &n->branch[i2].rect) )
					*area += IntersectedRectArea(&n->branch[i1].rect, &n->branch[i2].rect);

		for (int i=0; i<n->count; i++)
			checkNodeOverlap(n->branch[i].child, area);
	}
}


int comp(const void * t1,const void * t2) 
{
	int* a = (int*)t1;
	int* b = (int*)t2;

	if (*a==*b)
		return 0;
	else
		if (*a < *b)
			return -1;
		else
			return 1;
}
int comp_d0(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[0] == b->rect.boundary[0]) // x axis
		return 0;
	else
		if ( a->rect.boundary[0] < b->rect.boundary[0]) // x axis
			return -1;
		else
			return 1;
}
int comp_d1(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[1] == b->rect.boundary[1]) // y axis
		return 0;
	else
		if ( a->rect.boundary[1] < b->rect.boundary[1]) // y axis
			return -1;
		else
			return 1;
}
int comp_d2(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[2] == b->rect.boundary[2]) // z axis
		return 0;
	else
		if ( a->rect.boundary[2] < b->rect.boundary[2]) // z axis
			return -1;
		else
			return 1;
}
int comp_d3(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[3] == b->rect.boundary[3]) // z axis
		return 0;
	else
		if ( a->rect.boundary[3] < b->rect.boundary[3]) // z axis
			return -1;
		else
			return 1;
}
#if NUMDIMS > 4
int comp_d4(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[4] == b->rect.boundary[4]) // z axis
		return 0;
	else
		if ( a->rect.boundary[4] < b->rect.boundary[4]) // z axis
			return -1;
		else
			return 1;
}int comp_d5(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[5] == b->rect.boundary[5]) // z axis
		return 0;
	else
		if ( a->rect.boundary[5] < b->rect.boundary[5]) // z axis
			return -1;
		else
			return 1;
}int comp_d6(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[6] == b->rect.boundary[6]) // z axis
		return 0;
	else
		if ( a->rect.boundary[6] < b->rect.boundary[6]) // z axis
			return -1;
		else
			return 1;
}int comp_d7(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[7] == b->rect.boundary[7]) // z axis
		return 0;
	else
		if ( a->rect.boundary[7] < b->rect.boundary[7]) // z axis
			return -1;
		else
			return 1;
}int comp_d8(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[8] == b->rect.boundary[8]) // z axis
		return 0;
	else
		if ( a->rect.boundary[8] < b->rect.boundary[8]) // z axis
			return -1;
		else
			return 1;
}int comp_d9(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[9] == b->rect.boundary[9]) // z axis
		return 0;
	else
		if ( a->rect.boundary[9] < b->rect.boundary[9]) // z axis
			return -1;
		else
			return 1;
}int comp_d10(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[10] == b->rect.boundary[10]) // z axis
		return 0;
	else
		if ( a->rect.boundary[10] < b->rect.boundary[10]) // z axis
			return -1;
		else
			return 1;
}int comp_d11(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[11] == b->rect.boundary[11]) // z axis
		return 0;
	else
		if ( a->rect.boundary[11] < b->rect.boundary[11]) // z axis
			return -1;
		else
			return 1;
}int comp_d12(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[12] == b->rect.boundary[12]) // z axis
		return 0;
	else
		if ( a->rect.boundary[12] < b->rect.boundary[12]) // z axis
			return -1;
		else
			return 1;
}int comp_d13(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[13] == b->rect.boundary[13]) // z axis
		return 0;
	else
		if ( a->rect.boundary[13] < b->rect.boundary[13]) // z axis
			return -1;
		else
			return 1;
}int comp_d14(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[14] == b->rect.boundary[14]) // z axis
		return 0;
	else
		if ( a->rect.boundary[14] < b->rect.boundary[14]) // z axis
			return -1;
		else
			return 1;
}int comp_d15(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[15] == b->rect.boundary[15]) // z axis
		return 0;
	else
		if ( a->rect.boundary[15] < b->rect.boundary[15]) // z axis
			return -1;
		else
			return 1;
}
#if NUMDIMS > 16
int comp_d16(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[16] == b->rect.boundary[16]) // z axis
		return 0;
	else
		if ( a->rect.boundary[16] < b->rect.boundary[16]) // z axis
			return -1;
		else
			return 1;
}int comp_d17(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[17] == b->rect.boundary[17]) // z axis
		return 0;
	else
		if ( a->rect.boundary[17] < b->rect.boundary[17]) // z axis
			return -1;
		else
			return 1;
}int comp_d18(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[18] == b->rect.boundary[18]) // z axis
		return 0;
	else
		if ( a->rect.boundary[18] < b->rect.boundary[18]) // z axis
			return -1;
		else
			return 1;
}int comp_d19(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[19] == b->rect.boundary[19]) // z axis
		return 0;
	else
		if ( a->rect.boundary[19] < b->rect.boundary[19]) // z axis
			return -1;
		else
			return 1;
}int comp_d20(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[20] == b->rect.boundary[20]) // z axis
		return 0;
	else
		if ( a->rect.boundary[20] < b->rect.boundary[20]) // z axis
			return -1;
		else
			return 1;
}
int comp_d21(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[21] == b->rect.boundary[21]) // z axis
		return 0;
	else
		if ( a->rect.boundary[21] < b->rect.boundary[21]) // z axis
			return -1;
		else
			return 1;
}int comp_d22(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[22] == b->rect.boundary[22]) // z axis
		return 0;
	else
		if ( a->rect.boundary[22] < b->rect.boundary[22]) // z axis
			return -1;
		else
			return 1;
}int comp_d23(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[23] == b->rect.boundary[23]) // z axis
		return 0;
	else
		if ( a->rect.boundary[23] < b->rect.boundary[23]) // z axis
			return -1;
		else
			return 1;
}int comp_d24(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[24] == b->rect.boundary[24]) // z axis
		return 0;
	else
		if ( a->rect.boundary[24] < b->rect.boundary[24]) // z axis
			return -1;
		else
			return 1;
}int comp_d25(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[25] == b->rect.boundary[25]) // z axis
		return 0;
	else
		if ( a->rect.boundary[25] < b->rect.boundary[25]) // z axis
			return -1;
		else
			return 1;
}int comp_d26(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[26] == b->rect.boundary[26]) // z axis
		return 0;
	else
		if ( a->rect.boundary[26] < b->rect.boundary[26]) // z axis
			return -1;
		else
			return 1;
}int comp_d27(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[27] == b->rect.boundary[27]) // z axis
		return 0;
	else
		if ( a->rect.boundary[27] < b->rect.boundary[27]) // z axis
			return -1;
		else
			return 1;
}int comp_d28(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[28] == b->rect.boundary[28]) // z axis
		return 0;
	else
		if ( a->rect.boundary[28] < b->rect.boundary[28]) // z axis
			return -1;
		else
			return 1;
}int comp_d29(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[29] == b->rect.boundary[29]) // z axis
		return 0;
	else
		if ( a->rect.boundary[29] < b->rect.boundary[29]) // z axis
			return -1;
		else
			return 1;
}int comp_d30(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[30] == b->rect.boundary[30]) // z axis
		return 0;
	else
		if ( a->rect.boundary[30] < b->rect.boundary[30]) // z axis
			return -1;
		else
			return 1;
}int comp_d31(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[31] == b->rect.boundary[31]) // z axis
		return 0;
	else
		if ( a->rect.boundary[31] < b->rect.boundary[31]) // z axis
			return -1;
		else
			return 1;
}int comp_d32(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[32] == b->rect.boundary[32]) // z axis
		return 0;
	else
		if ( a->rect.boundary[32] < b->rect.boundary[32]) // z axis
			return -1;
		else
			return 1;
}int comp_d33(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[33] == b->rect.boundary[33]) // z axis
		return 0;
	else
		if ( a->rect.boundary[33] < b->rect.boundary[33]) // z axis
			return -1;
		else
			return 1;
}int comp_d34(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[34] == b->rect.boundary[34]) // z axis
		return 0;
	else
		if ( a->rect.boundary[34] < b->rect.boundary[34]) // z axis
			return -1;
		else
			return 1;
}int comp_d35(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[35] == b->rect.boundary[35]) // z axis
		return 0;
	else
		if ( a->rect.boundary[35] < b->rect.boundary[35]) // z axis
			return -1;
		else
			return 1;
}int comp_d36(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[36] == b->rect.boundary[36]) // z axis
		return 0;
	else
		if ( a->rect.boundary[36] < b->rect.boundary[36]) // z axis
			return -1;
		else
			return 1;
}int comp_d37(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[37] == b->rect.boundary[37]) // z axis
		return 0;
	else
		if ( a->rect.boundary[37] < b->rect.boundary[37]) // z axis
			return -1;
		else
			return 1;
}int comp_d38(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[38] == b->rect.boundary[38]) // z axis
		return 0;
	else
		if ( a->rect.boundary[38] < b->rect.boundary[38]) // z axis
			return -1;
		else
			return 1;
}int comp_d39(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[39] == b->rect.boundary[39]) // z axis
		return 0;
	else
		if ( a->rect.boundary[39] < b->rect.boundary[39]) // z axis
			return -1;
		else
			return 1;
}int comp_d40(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[40] == b->rect.boundary[40]) // z axis
		return 0;
	else
		if ( a->rect.boundary[40] < b->rect.boundary[40]) // z axis
			return -1;
		else
			return 1;
}int comp_d41(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[41] == b->rect.boundary[41]) // z axis
		return 0;
	else
		if ( a->rect.boundary[41] < b->rect.boundary[41]) // z axis
			return -1;
		else
			return 1;
}int comp_d42(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[42] == b->rect.boundary[42]) // z axis
		return 0;
	else
		if ( a->rect.boundary[42] < b->rect.boundary[42]) // z axis
			return -1;
		else
			return 1;
}int comp_d43(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[43] == b->rect.boundary[43]) // z axis
		return 0;
	else
		if ( a->rect.boundary[43] < b->rect.boundary[43]) // z axis
			return -1;
		else
			return 1;
}int comp_d44(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[44] == b->rect.boundary[44]) // z axis
		return 0;
	else
		if ( a->rect.boundary[44] < b->rect.boundary[44]) // z axis
			return -1;
		else
			return 1;
}int comp_d45(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[45] == b->rect.boundary[45]) // z axis
		return 0;
	else
		if ( a->rect.boundary[45] < b->rect.boundary[45]) // z axis
			return -1;
		else
			return 1;
}int comp_d46(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[46] == b->rect.boundary[46]) // z axis
		return 0;
	else
		if ( a->rect.boundary[46] < b->rect.boundary[46]) // z axis
			return -1;
		else
			return 1;
}int comp_d47(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[47] == b->rect.boundary[47]) // z axis
		return 0;
	else
		if ( a->rect.boundary[47] < b->rect.boundary[47]) // z axis
			return -1;
		else
			return 1;
}int comp_d48(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[48] == b->rect.boundary[48]) // z axis
		return 0;
	else
		if ( a->rect.boundary[48] < b->rect.boundary[48]) // z axis
			return -1;
		else
			return 1;
}int comp_d49(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[49] == b->rect.boundary[49]) // z axis
		return 0;
	else
		if ( a->rect.boundary[49] < b->rect.boundary[49]) // z axis
			return -1;
		else
			return 1;
}int comp_d50(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[50] == b->rect.boundary[50]) // z axis
		return 0;
	else
		if ( a->rect.boundary[50] < b->rect.boundary[50]) // z axis
			return -1;
		else
			return 1;
}int comp_d51(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[51] == b->rect.boundary[51]) // z axis
		return 0;
	else
		if ( a->rect.boundary[51] < b->rect.boundary[51]) // z axis
			return -1;
		else
			return 1;
}int comp_d52(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[52] == b->rect.boundary[52]) // z axis
		return 0;
	else
		if ( a->rect.boundary[52] < b->rect.boundary[52]) // z axis
			return -1;
		else
			return 1;
}int comp_d53(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[53] == b->rect.boundary[53]) // z axis
		return 0;
	else
		if ( a->rect.boundary[53] < b->rect.boundary[53]) // z axis
			return -1;
		else
			return 1;
}
int comp_d54(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[54] == b->rect.boundary[54]) // z axis
		return 0;
	else
		if ( a->rect.boundary[54] < b->rect.boundary[54]) // z axis
			return -1;
		else
			return 1;
}int comp_d55(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[55] == b->rect.boundary[55]) // z axis
		return 0;
	else
		if ( a->rect.boundary[55] < b->rect.boundary[55]) // z axis
			return -1;
		else
			return 1;
}int comp_d56(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[56] == b->rect.boundary[56]) // z axis
		return 0;
	else
		if ( a->rect.boundary[56] < b->rect.boundary[56]) // z axis
			return -1;
		else
			return 1;
}int comp_d57(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[57] == b->rect.boundary[57]) // z axis
		return 0;
	else
		if ( a->rect.boundary[57] < b->rect.boundary[57]) // z axis
			return -1;
		else
			return 1;
}int comp_d58(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[58] == b->rect.boundary[58]) // z axis
		return 0;
	else
		if ( a->rect.boundary[58] < b->rect.boundary[58]) // z axis
			return -1;
		else
			return 1;
}int comp_d59(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[59] == b->rect.boundary[59]) // z axis
		return 0;
	else
		if ( a->rect.boundary[59] < b->rect.boundary[59]) // z axis
			return -1;
		else
			return 1;
}int comp_d60(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[60] == b->rect.boundary[60]) // z axis
		return 0;
	else
		if ( a->rect.boundary[60] < b->rect.boundary[60]) // z axis
			return -1;
		else
			return 1;
}int comp_d61(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[61] == b->rect.boundary[61]) // z axis
		return 0;
	else
		if ( a->rect.boundary[61] < b->rect.boundary[61]) // z axis
			return -1;
		else
			return 1;
}int comp_d62(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[62] == b->rect.boundary[62]) // z axis
		return 0;
	else
		if ( a->rect.boundary[62] < b->rect.boundary[62]) // z axis
			return -1;
		else
			return 1;
}int comp_d63(const void * t1,const void * t2) 
{
	BVH_Branch* a = (BVH_Branch*)t1;
	BVH_Branch* b = (BVH_Branch*)t2;

	if ( a->rect.boundary[63] == b->rect.boundary[63]) // z axis
		return 0;
	else
		if ( a->rect.boundary[63] < b->rect.boundary[63]) // z axis
			return -1;
		else
			return 1;
}
#endif
#endif


__global__ void globalSetDeviceRoot(char* buf, int partition_no, int NUMBLOCK, int PARTITIONED )
{ 

	if(partition_no==0){
		devNUMBLOCK = NUMBLOCK;
		deviceRoot = (struct Node**) malloc( sizeof(struct Node*) * PARTITIONED );
	}
	deviceRoot[partition_no] = (struct Node*) buf;
}
__global__ void globalSetDeviceBVHRoot(char* buf, int partition_no, int NUMBLOCK, int PARTITIONED )
{ 

	if(partition_no==0){
		devNUMBLOCK = NUMBLOCK;
		deviceBVHRoot = (BVH_Node**) malloc( sizeof(BVH_Node*) * PARTITIONED );
	}
	deviceBVHRoot[partition_no] = (BVH_Node*) buf;
}
__global__ void globalFreeDeviceRoot(int PARTITIONED )
{ 

	for(int i=0; i<PARTITIONED; i++)
		free(deviceRoot[i]);
//	free(deviceRoot);
}
__global__ void globalFreeDeviceBVHRoot(int PARTITIONED )
{ 
	for(int i=0; i<PARTITIONED; i++)
		free(deviceBVHRoot[i]);
//	free(deviceBVHRoot);
}


#if NUMDIMS < 64
__global__ void globaltranspose_node(int partition_no, int totalNodes)
{
	int tid = threadIdx.x;

	__shared__ struct Node_SOA node_soa;
	__shared__ char* node_ptr;
	__shared__ char* node_soa_ptr;

	node_ptr = (char*) deviceRoot[partition_no];
	node_soa_ptr = (char*)&node_soa;

	for( int i=0; i< totalNodes; i++ )
	{

		//memcpy a node to node_soa_ptr
		//transpose node -> node_SOA

		for( int d=0; d<NUMDIMS*2; d++)
			memcpy(node_soa_ptr+(tid*4)+(NODECARD*4)*d, node_ptr+8+(sizeof(struct Branch)*tid)+(d*4), sizeof(float)); 															//copy boundary
		memcpy(node_soa_ptr+(8*NODECARD*NUMDIMS)+tid*4, node_ptr+8+(sizeof(struct Branch)*tid)+sizeof(struct Rect), sizeof(int)); 								//copy index code
		memcpy(node_soa_ptr+(8*NODECARD*NUMDIMS)+(NODECARD*4)+tid*8, node_ptr+8+(sizeof(struct Branch)*tid)+sizeof(struct Rect)+8, sizeof(long)); //copy child pointer

		if( tid == 0 )
			memcpy(node_soa_ptr+(8*NODECARD*NUMDIMS)+(NODECARD*4)+(NODECARD*8), node_ptr, sizeof(int)*2); 																						//copy count and level
		__syncthreads();
		//and then copy to deviceRoot again
		//memcpy(node_ptr, node_soa_ptr, PGSIZE);	
		//writing global memory in parallel may cause a program
		for( int d=0; d<NUMDIMS*2; d++)
			memcpy(node_ptr+(tid*4)+(NODECARD*4)*d, node_soa_ptr+(tid*4)+(NODECARD*4)*d, sizeof(float));
		memcpy(node_ptr+(8*NODECARD*NUMDIMS)+tid*4, node_soa_ptr+(8*NODECARD*NUMDIMS)+tid*4, sizeof(int));
		memcpy(node_ptr+(8*NODECARD*NUMDIMS)+(NODECARD*4)+tid*8, node_soa_ptr+(8*NODECARD*NUMDIMS)+(NODECARD*4)+tid*8, sizeof(long));

		if( tid == 0 )
			memcpy(node_ptr+(8*NODECARD*NUMDIMS)+(NODECARD*4)+(NODECARD*8), node_soa_ptr+(8*NODECARD*NUMDIMS)+(NODECARD*4)+(NODECARD*8), sizeof(int)*2);
		__syncthreads();

		if( tid == 0)
			node_ptr += PGSIZE;
		__syncthreads();
	}
}

__global__ void globaltranspose_BVHnode(int partition_no, int totalNodes)
{
	int tid = threadIdx.x;

	__shared__ struct BVH_Node_SOA node_soa;
	__shared__ char* node_ptr;
	__shared__ char* node_soa_ptr;

	node_ptr = (char*) deviceBVHRoot[partition_no];
	node_soa_ptr = (char*)&node_soa;
	for( int i=0; i< totalNodes; i++ )
	{

		//memcpy a node to node_soa_ptr
		//transpose node -> node_SOA
		for( int d=0; d<NUMDIMS*2; d++)
			memcpy(node_soa_ptr+(tid*4)+(NODECARD*4)*d, node_ptr+8+(sizeof(BVH_Branch)*tid)+(d*4), sizeof(float)); 															//copy boundary
		memcpy(node_soa_ptr+(8*NODECARD*NUMDIMS)+tid*4, node_ptr+8+(sizeof(BVH_Branch)*tid)+sizeof(struct Rect), sizeof(int)); 								//copy index code
		memcpy(node_soa_ptr+(8*NODECARD*NUMDIMS)+(NODECARD*4)+tid*8, node_ptr+8+(sizeof(BVH_Branch)*tid)+sizeof(struct Rect)+8, sizeof(long)); //copy child pointer

		if( tid == 0 )
		{
			//copy count, level, parent and sibling pointer
			memcpy(node_soa_ptr+(8*NODECARD*NUMDIMS)+(NODECARD*4)+(NODECARD*8), node_ptr, sizeof(int)*2);
			memcpy(node_soa_ptr+(8*NODECARD*NUMDIMS)+(NODECARD*4)+(NODECARD*8)+8, node_ptr+8+sizeof(BVH_Branch)*NODECARD,sizeof(long)*2);
		}
		__syncthreads();

		//and then copy to deviceRoot again
		for( int d=0; d<NUMDIMS*2; d++)
			memcpy(node_ptr+(tid*4)+(NODECARD*4)*d, node_soa_ptr+(tid*4)+(NODECARD*4)*d, sizeof(float));
		memcpy(node_ptr+(8*NODECARD*NUMDIMS)+tid*4, node_soa_ptr+(8*NODECARD*NUMDIMS)+tid*4, sizeof(int));
		memcpy(node_ptr+(8*NODECARD*NUMDIMS)+(NODECARD*4)+tid*8, node_soa_ptr+(8*NODECARD*NUMDIMS)+(NODECARD*4)+tid*8, sizeof(long));

		if( tid == 0)
		memcpy(node_ptr+(8*NODECARD*NUMDIMS)+(NODECARD*4)+(NODECARD*8), node_soa_ptr+(8*NODECARD*NUMDIMS)+(NODECARD*4)+(NODECARD*8), (sizeof(int)*2)+(sizeof(long)*2));
		__syncthreads();

		if( tid == 0)
			node_ptr += BVH_PGSIZE;
		__syncthreads();

	}
}

#else

__global__ void globaltranspose_node(int partition_no, int totalNodes)
{
	struct Node_SOA node_soa;
	struct Node * node_ptr = (struct Node*) deviceRoot[partition_no];

	for( int i=0; i< totalNodes; i++ )
	{

		//memcpy a node to node_soa_ptr
		//transpose node -> node_SOA

		for(int b=0; b<NODECARD; b++)
		{
			for( int d=0; d<NUMDIMS*2; d++)
			{
				node_soa.boundary[d*NODECARD+b] = node_ptr->branch[b].rect.boundary[d];
			}

			node_soa.index[b] = (int) node_ptr->branch[b].hIndex;
			char* tmp = (char*) node_ptr;
			memcpy(&node_soa.child[b], tmp+8+(b*sizeof(struct Branch))+sizeof(struct Rect)+8, sizeof(long));
		}
		node_soa.count = node_ptr->count;
		node_soa.level = node_ptr->level;

		memcpy(node_ptr, &node_soa, PGSIZE);

		node_ptr++;
	}
}

__global__ void globaltranspose_BVHnode(int partition_no, int totalNodes)
{
	BVH_Node_SOA node_soa;
	BVH_Node * node_ptr = (BVH_Node*) deviceBVHRoot[partition_no];

	for( int i=0; i< totalNodes; i++ )
	{

		//memcpy a node to node_soa_ptr
		//transpose node -> node_SOA

		for(int b=0; b<NODECARD; b++)
		{
			for( int d=0; d<NUMDIMS*2; d++)
			{
				node_soa.boundary[d*NODECARD+b] = node_ptr->branch[b].rect.boundary[d];
			}

			node_soa.index[b] = (int) node_ptr->branch[b].mortonCode;
			char* tmp = (char*) node_ptr;
			memcpy(&node_soa.child[b], tmp+8+(b*sizeof(BVH_Branch))+sizeof(struct Rect)+8, sizeof(long));
		}
		node_soa.count = node_ptr->count;
		node_soa.level = node_ptr->level;

		node_soa.parent = (BVH_Node_SOA*) node_ptr->parent;
		node_soa.sibling = (BVH_Node_SOA*) node_ptr->sibling;

		memcpy(node_ptr, &node_soa, BVH_PGSIZE);

		node_ptr++;
	}
}

#endif


//	long nil_ptr = 0x0;
		/*
		if( tid == 0 )
		{
			struct Node* node = (struct Node*)node_ptr;
			printf("POINT1\n");

			for(int n=0; n<NODECARD; n++)
				memcpy(&node->branch[n].child, &nil_ptr ,sizeof(long) );

			for(int d=0; d<NUMDIMS*2; d++)
			{
				for(int n=0; n<node->count; n++)
					printf("%.6f \n", node->branch[n].rect.boundary[d]);
			}
			for(int n=0; n<node->count; n++)
				printf("%d \n", node->branch[n].hIndex);
			for(int n=0; n<node->count; n++)
			printf("%lu \n", node->branch[n].child);
		}
		__syncthreads();
		*/
		/*
		if( tid == 0)
		{
			printf("POINT2\n");
		for( int n=0; n<NODECARD*2*NUMDIMS; n++)
			printf("%.6f \n", node_soa.boundary[n]);

		for( int n=0; n<NODECARD; n++)
			printf("%d \n", node_soa.index[n]);

		for( int n=0; n<NODECARD; n++)
			printf("%lu \n", node_soa.child[n]);

		printf("%d %d\n", node_soa.count, node_soa.level);
		}
		*/


__global__ void globalDesignTraversalScenario()
{
	int tid = threadIdx.x;

	struct Node_SOA* root = (struct Node_SOA*) deviceRoot[0];
	struct Node_SOA* node = root;
	
	for( int i=0; i<129; i++)
	{
		for(  int d=0; d<NUMDIMS; d++)
		{
			node->boundary[tid+(d*NODECARD)] = 1.1f;
			node->boundary[tid+((d+NUMDIMS)*NODECARD)] = -0.1f;
		}
		
		if( i == 0 )// root node
		{
			if( tid == 0)
			{
				for(int d=0; d<NUMDIMS; d++)
				{
					node->boundary[2+(d*NODECARD)] = 0.0f;
					node->boundary[2+((d+NUMDIMS)*NODECARD)] = 1.0f;
					node->boundary[119+(d*NODECARD)] = 0.0f;
					node->boundary[119+((d+NUMDIMS)*NODECARD)] = 1.0f;
				}
			}
			__syncthreads();
		}
		else if( i == 3 )
		{
			if( tid == 0)
			{
				for(int d=0; d<NUMDIMS; d++)
				{
					node->boundary[(d*NODECARD)] = 0.0f;
					node->boundary[((d+NUMDIMS)*NODECARD)] = 1.0f;
				}
			}
			__syncthreads();
		}
		else if( i == 120 )
		{
			if( tid == 0)
			{
				for(int d=0; d<NUMDIMS; d++)
				{
					node->boundary[(d*NODECARD)] = 0.0f;
					node->boundary[((d+NUMDIMS)*NODECARD)] = 1.0f;
				}
			}
		}

		node++;
	}

	for( int i=0; i<16384; i++)
	{
		for(int d=0; d<NUMDIMS; d++)
		{
			node->boundary[tid+(d*NODECARD)] = 0.0f;
			node->boundary[tid+((d+NUMDIMS)*NODECARD)] = 1.0f;
		}
		node++;
	}
}
__global__ void globalDesignTraversalScenarioBVH()
{
	int tid = threadIdx.x;

	BVH_Node_SOA* root = (BVH_Node_SOA*) deviceBVHRoot[0];
	BVH_Node_SOA* node = root;
	
	for( int i=0; i<129; i++)
	{
		for(  int d=0; d<NUMDIMS; d++)
		{
			node->boundary[tid+(d*NODECARD)] = 1.1f;
			node->boundary[tid+((d+NUMDIMS)*NODECARD)] = -0.1f;
		}
		
		if( i == 0 )// root node
		{
			if( tid == 0)
			{
				for(int d=0; d<NUMDIMS; d++)
				{
					node->boundary[2+(d*NODECARD)] = 0.0f;
					node->boundary[2+((d+NUMDIMS)*NODECARD)] = 1.0f;
					node->boundary[119+(d*NODECARD)] = 0.0f;
					node->boundary[119+((d+NUMDIMS)*NODECARD)] = 1.0f;
				}
			}
			__syncthreads();
		}
		else if( i == 3 )
		{
			if( tid == 0)
			{
				for(int d=0; d<NUMDIMS; d++)
				{
					node->boundary[(d*NODECARD)] = 0.0f;
					node->boundary[((d+NUMDIMS)*NODECARD)] = 1.0f;
				}
			}
			__syncthreads();
		}
		else if( i == 120 )
		{
			if( tid == 0)
			{
				for(int d=0; d<NUMDIMS; d++)
				{
					node->boundary[(d*NODECARD)] = 0.0f;
					node->boundary[((d+NUMDIMS)*NODECARD)] = 1.0f;
				}
			}
		}

		node++;
	}

	for( int i=0; i<16384; i++)
	{
		for(int d=0; d<NUMDIMS; d++)
		{
			node->boundary[tid+(d*NODECARD)] = 0.0f;
			node->boundary[tid+((d+NUMDIMS)*NODECARD)] = 1.0f;
		}
		node++;
	}
}

int find_an_available_gpu(int num_of_gpus)
{
	int i;
	for(i=0; i<num_of_gpus; i++) 
	{
		hipError_t error = hipSetDevice(i);

		if( error != hipSuccess)
			continue;

		size_t avail, total;
		hipMemGetInfo( &avail, &total );
		size_t used = total-avail;
		float per = ( (double)used/(double)total)*100;

		if( per <= 10.0 )
		{
			printf("%dth GPU is selected \n", i);
	    return i;
		}
		else if( i == 7 )
		{
			char hostname[10];
			gethostname(hostname,10);
			printf("Unfortunately, There is no available device on %s",hostname);
			return 0;
		}
	}
	return -1;
}

//convert and print decimal in binary from most significant bit
void printDecimalToBinary(unsigned long long num, int order)
{
	if( order < 64 ) 
	{
	  printDecimalToBinary(num>>1, ++order);

	if( num % 2 == 0 )
		printf("0");
	else
		printf("1");
	}
}

//convert and print decimal in binary from least significant bit
// num : num to print out in binary
// order : bit position to print now 
// nbits : number of bits to print 
// valid bits : number of valid bits from most significant bit
void printCommonPrefix(unsigned long long num, int order, int nbits, int valid_bits)
{
	if( order < nbits ) 
	{
		printCommonPrefix(num>>1, ++order, nbits, valid_bits);

		if(order <= (64-valid_bits))
		{
			printf("X");
		}
		else
		{
			if( num % 2 == 0 )
				printf("0");
			else
				printf("1");
		}
	}
}

unsigned long long RightShift(unsigned long long val, int shift)
{
	  int loop = shift/30;
		  int i;

			  for(i=0; i < loop ; i++)
					    val = val >> 30;

				  val = val >> shift%30;

					  return val;
}
unsigned long long LeftShift(unsigned long long val, int shift)
{
	  int loop = shift/30;
		  int i;

			  for(i=0; i < loop ; i++)
					    val = val << 30;

				  val = val << shift%30;

					  return val;
}

void ConvertHilbertIndexToBoundingBox(unsigned long long index, int X ,float* rect)
{
	int i,j;
	int nbits = X%NUMDIMS;
	unsigned long long coord[NUMDIMS];
	unsigned long long coords[4][NUMDIMS];
	//unsigned long long coords[nbits*2][NUMDIMS];

	unsigned long long current_num;
	unsigned long long start_num = LeftShift( RightShift(index ,X), X);
	unsigned long long increments = LeftShift(1 , (X/NUMDIMS)*NUMDIMS) ;
	unsigned long long end_num =  start_num | (LeftShift(1 , X) -1);

//	printf("1 end num : ");
//	printCommonPrefix(end_num, 0 , 64, 64);
//	printf("\n\n");

	end_num = LeftShift(RightShift( end_num, (X/NUMDIMS)*NUMDIMS), (X/NUMDIMS)*NUMDIMS);

//	printf("2 end num : ");
//	printCommonPrefix(end_num, 0 , 64, 64);
//	printf("\n\n");

//	printf("X : %d\n", X);

//	printf("index : ");
//	printCommonPrefix(index, 0 , 64, 64);
//	printf("\n\n");

	for(i=0, current_num = start_num; current_num <= end_num; current_num+=increments, i++)
	{
		if( i!=0 && current_num <= start_num) break;

		/*
		if( index ==  61763094443385092)
		{
			printf("index : ");
			printCommonPrefix(index, 0 , 64, 64);
			printf("\n\n");

			printf("cur num : ");
			printCommonPrefix(current_num, 0 , 64, 64);
			printf("\n\n");

			printf("inc num : ");
			printCommonPrefix(increments, 0 , 64, 64);
			printf("\n\n");

			printf("end num : ");
			printCommonPrefix(end_num, 0 , 64, 64);
			printf("\n\n");
		}
		*/

//		printf("%d\n",__LINE__);
		hilbert_i2c(NUMDIMS, 20, current_num, coords[i]);
		for(j=0; j<NUMDIMS; j++)
		{
//		  printf("%d\n",__LINE__);
			coords[i][j] = LeftShift(RightShift(coords[i][j] , (X/NUMDIMS)) , (X/NUMDIMS));

			/*
			if( index ==  61763094443385092)
			{
				printf("coords[%d][%d] %llu\n", i,j,coords[i][j]);
			}
			*/
		}
	}

  hilbert_i2c(NUMDIMS, 20, index, coord);

	int diff_bits[NUMDIMS];
	for(i=0; i<NUMDIMS; i++)
	{
//		printf("%d\n",__LINE__);
		diff_bits[i]=0;
		if( nbits == 1 )
		diff_bits[i] = __builtin_popcountll( coords[0][i] ^ coords[1][i]);
		else if( nbits == 2 )
		diff_bits[i] = __builtin_popcountll( (coords[0][i]^coords[1][i])|(coords[0][i]^coords[2][i])|(coords[0][i]^coords[3][i]) );
		// TO DO :: nbits 3, 4 ~~

		diff_bits[i] += X/NUMDIMS;

		/*
		printf("org coord[%d] %llu \n", i, coord[i]);
		printCommonPrefix(coord[i] , 0 , 64, 64);
		printf("\n\n");
		*/

		coord[i] = LeftShift( RightShift( coord[i] , diff_bits[i]), diff_bits[i]);
		//printf("min coord[%d] %llu \n", i, coord[i]);
		//printCommonPrefix(coord[i] , 0 , 64, 64);
		//printf("\n");
		rect[i] = coord[i]/1000000.0f;
		//printf("min rect[%d] %f \n\n", i, rect[i]);

		coord[i] += (LeftShift( 1 , diff_bits[i] ) -1 );
		//printf("max coord[%d] %llu \n", i, coord[i]);
		//printCommonPrefix(coord[i] , 0 , 64, 64);
		//printf("\n");
		rect[i+NUMDIMS] = coord[i]/1000000.0f;
		//printf("max rect[%d] %f \n\n", i+NUMDIMS, rect[i+NUMDIMS]);

	}

}

